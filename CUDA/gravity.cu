
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <signal.h>
#define fg 0.1
#define fe 100
#define damp 0.01
#define critical_factor  0.008
#define SCREEN_WIDTH 600
#define SCREEN_HEIGHT 600
const int MAX_SIZE = 500;
// [self.x,self.y,self.vx,self.vy,self.m,self.charge]
//    0     1       2       3       4       5
__global__ void myop( float * d_output, float *d_input,int size)
{
  __shared__ float buffer[MAX_SIZE*6];
  int id = threadIdx.x;
  float my[6];
  for(int i = 0; i< 6; i++)
  {
    float temp  =d_input[6*id+i];
  buffer[6*id+i]=temp;
  my[i]=temp;

  }
  __syncthreads();

  for(int i = 0; i< size; i++)
  {
    if(i==id)
    continue;
    float other[6];
    for(int j = 0; j<6;j++)
    {
      other[j]=buffer[6*i+j];
    }
    float dx = my[0]-other[0];
    float dy = my[1]-other[1];
    float r = sqrtf(dx*dx+dy*dy);
    float fx =-(fg*my[4]*other[4]-fe*my[5]*other[5])*dx;
    fx=fx/(r*r*r);
    float fy =-(fg*my[4]*other[4]-fe*my[5]*other[5])*dy;
    fy=fy/(r*r*r);
    if(r>10)
    {
      my[2]+=(fx/my[4]);
      my[3]+=(fy/my[4]);
    }
    else if(r>3)
    {
      my[2]+=fx*critical_factor*r*r;
      my[3]+=fy*critical_factor*r*r;
      my[2]-=damp*(my[2]-other[2])*abs(my[2])*abs(my[5]-other[5])*abs(my[1]-other[1]);
      my[3]-=damp*(my[3]-other[3])*abs(my[3])*abs(my[5]-other[5])*abs(my[2]-other[2]);
    }

  }
  buffer[6*id+2]=my[2];
  buffer[6*id+3]=my[3];
  buffer[6*id]+=my[2];
  buffer[6*id+1]+=my[3];
  if(buffer[6*id]>SCREEN_WIDTH)
  {
    buffer[6*id+2]=-buffer[6*id+2];
    buffer[6*id]=SCREEN_WIDTH-2;
  }
  else if(buffer[6*id]<0)
  {
    buffer[6*id+2]=-buffer[6*id+2];
    buffer[6*id]=2;
  }
  if(buffer[6*id+1]>SCREEN_HEIGHT)
  {
    buffer[6*id+3]=-buffer[6*id+3];
    buffer[6*id+1]=SCREEN_HEIGHT-2;
  }
  else if(buffer[6*id+1]<0)
    {
      buffer[6*id+3]=-buffer[6*id+3];
      buffer[6*id+1]=2;
    }
  __syncthreads();
  for(int i = 0; i< 6; i++)
  {
    d_output[6*id+i]=buffer[6*id+i];
  }
}
extern "C"{
float * calcAndUpdate(float * input, int size)
{
  float * d_input, * d_output;
  float *result = (float *)malloc((size*6+2)*sizeof(float));
  if(size>500)
  {
    return 0;

  }
  hipMalloc(&d_input,(size*6)*sizeof(float));
  hipMalloc(&d_output,(size*6)*sizeof(float));
  hipMemcpy(d_input,input,(size*6)*sizeof(float),hipMemcpyHostToDevice);
  myop<<<1,size>>>(d_output,d_input,size);
  hipMemcpy(result,d_output,(size*6)*sizeof(float),hipMemcpyDeviceToHost);
  // for(int i = 0; i< 6*size; i++)
  // {
  //   printf("%f \t",result[i]);
  //   if(i%6==0)
  //   {
  //     printf("\n");
  //   }
  // }
  return result;

}
}
int main()
{

}
