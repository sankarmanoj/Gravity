
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#define fg 0.00001
#define fe 0.001
#define damp 0.0001
#define critical_factor  0
#define SCREEN_WIDTH 600
#define SCREEN_HEIGHT 600
#define STRONG_FORCE 0.001
const int BLOCK_SIZE = 1024;
// [self.x,self.y,self.z,self.vx,self.vy,self.vz,self.m,self.charge]
//    0     1       2       3       4       5     6         7
__global__ void myop( float *d_input)
{
  __shared__ float buffer[BLOCK_SIZE*8];
  int myid = threadIdx.x+BLOCK_SIZE*blockIdx.x;
  int otherid = threadIdx.x +BLOCK_SIZE*blockIdx.y;
  // printf("myid=%d, otherid=%d, threadIdx=%d, blockX=%d, blockY=%d \n",myid,otherid,threadIdx.x,blockIdx.x,blockIdx.y);
  float my[8];
  for(int i = 0; i< 8; i++)
  {
    my[i]=d_input[8*myid+i];
    buffer[8*threadIdx.x + i]=d_input[8*otherid+i];
    }
  __syncthreads();


  for(int i = 0; i< BLOCK_SIZE; i++)
  {
    if(blockIdx.x==blockIdx.y && threadIdx.x==i)
    continue;
    float other[8];
    for(int j = 0; j<8;j++)
    {
    //      printf("%f[%d],(%d,%d) \n",buffer[8*i+j],8*i+j,myid,blockIdx.y);
      other[j]=buffer[8*i+j];
    }
    float dx = my[0]-other[0];
    float dy = my[1]-other[1];
    float dz = my[2]-other[2];
    float r = sqrtf(dx*dx+dy*dy+dz*dz);
    float fx =-(fg*my[6]*other[6]-fe*my[7]*other[7])*dx;
    fx=fx/(r*r*r);
    float fy =-(fg*my[6]*other[6]-fe*my[7]*other[7])*dy;
    fy=fy/(r*r*r);
    float fz =-(fg*my[6]*other[6]-fe*my[7]*other[7])*dz;
    fz=fz/(r*r*r);
    if(fx!=fx || fy!=fy||isnan(fy))
    continue;
    if(r>0.2f)
    {
      my[3]+=(fx/my[6]);
      my[4]+=(fy/my[6]);
      my[5]+=(fz/my[6]);
    }
    else if(r<0.2f&r>0.01f)
    {
      if((my[7]+other[7])>1.9f)
      {
        my[3]-=(dx/r)*STRONG_FORCE;
        my[4]-=(dy/r)*STRONG_FORCE;
        my[5]-=(dz/r)*STRONG_FORCE;
        my[3]-=(my[3]-other[3])*damp;
        my[4]-=(my[4]-other[4])*damp;
        my[5]-=(my[5]-other[5])*damp;
      }
    }

  }


my[3]-=d_input[8*myid+3];
my[4]-=d_input[8*myid+4];
my[5]-=d_input[8*myid+5];
  for(int i = 3; i< 6; i++)
  {

    atomicAdd(&d_input[8*myid+i],my[i]);
  }
  __syncthreads();
  if(blockIdx.y==0)
  {
    for(int i = 0; i< 3; i++)
    {
      d_input[8*myid+i]+=d_input[8*myid+i+3];
    }

  }


}
float * calcAndUpdate(float * input, int size)
{
  float * d_input, * d_output;
  float *result = (float *)malloc((size*8)*sizeof(float));
  if(!(size%BLOCK_SIZE==0))
  {
    printf("Size is not a multiple of %d",BLOCK_SIZE);
    exit(0);
  }
  hipMalloc(&d_input,(size*8)*sizeof(float));
  hipMalloc(&d_output,(size*8)*sizeof(float));
//  printf("cuda Size = %lu",(size*8)*sizeof(float));
  hipMemcpy(d_input,input,(size*8)*sizeof(float),hipMemcpyHostToDevice);
  int numberOfBlocks = size/BLOCK_SIZE;
  dim3 blocks(numberOfBlocks,numberOfBlocks);
  myop<<<blocks,BLOCK_SIZE>>>(d_input);
  hipError_t errSync  = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess)
    printf("Sync kernel error: %sn", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %sn", hipGetErrorString(errAsync));
  hipMemcpy(result,d_input,(size*8)*sizeof(float),hipMemcpyDeviceToHost);
  free(input);
  return result;

}
