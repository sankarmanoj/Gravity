
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#define fg 0.0000001
#define fe 0.0001
#define damp 0.0001
#define critical_factor  0
#define SCREEN_WIDTH 600
#define SCREEN_HEIGHT 600
#define STRONG_FORCE 0.001
const int MAX_SIZE = 1023;
// [self.x,self.y,self.z,self.vx,self.vy,self.vz,self.m,self.charge]
//    0     1       2       3       4       5     6         7
__global__ void myop( float * d_output, float *d_input,int size)
{
  __shared__ float buffer[MAX_SIZE*8];
  int id = threadIdx.x;
  float my[8];
  for(int i = 0; i< 8; i++)
  {
    float temp  =d_input[8*id+i];
  buffer[8*id+i]=temp;
  my[i]=temp;

  }
  __syncthreads();

  for(int i = 0; i< size; i++)
  {
    if(i==id)
    continue;
    float other[8];
    for(int j = 0; j<8;j++)
    {
      other[j]=buffer[8*i+j];
    }
    float dx = my[0]-other[0];
    float dy = my[1]-other[1];
    float dz = my[2]-other[2];
    float r = sqrtf(dx*dx+dy*dy+dz*dz);

    float fx =-(fg*my[6]*other[6]-fe*my[7]*other[7])*dx;
    fx=fx/(r*r*r);
    float fy =-(fg*my[6]*other[6]-fe*my[7]*other[7])*dy;
    fy=fy/(r*r*r);
    float fz =-(fg*my[6]*other[6]-fe*my[7]*other[7])*dz;
    fz=fz/(r*r*r);
    if(fx!=fx || fy!=fy||isnan(fy))
    continue;
    if(r>0.2f)
    {
      my[3]+=(fx/my[6]);
      my[4]+=(fy/my[6]);
      my[5]+=(fz/my[6]);
    }
    else if(r<0.2f&r>0.01f)
    {
      if((my[7]+other[7])>1.9f)
      {
        my[3]-=(dx/r)*STRONG_FORCE;
        my[4]-=(dy/r)*STRONG_FORCE;
        my[5]-=(dz/r)*STRONG_FORCE;
        my[3]-=(my[3]-other[3])*damp;
        my[4]-=(my[4]-other[4])*damp;
        my[5]-=(my[5]-other[5])*damp;
      }
    }

  }


  buffer[8*id+3]=my[3];
  buffer[8*id+4]=my[4];
  buffer[8*id+5]=my[5];
  buffer[8*id]+=my[3];
  buffer[8*id+1]+=my[4];
  buffer[8*id+2]+=my[5];
  __syncthreads();
  for(int i = 0; i< 8; i++)
  {
    d_output[8*id+i]=buffer[8*id+i];
  }

}
float * calcAndUpdate(float * input, int size)
{
  float * d_input, * d_output;
  float *result = (float *)malloc((size*8)*sizeof(float));
  if(size>MAX_SIZE)
  {
    return 0;

  }
  hipMalloc(&d_input,(size*8)*sizeof(float));
  hipMalloc(&d_output,(size*8)*sizeof(float));
//  printf("cuda Size = %lu",(size*8)*sizeof(float));
  hipMemcpy(d_input,input,(size*8)*sizeof(float),hipMemcpyHostToDevice);
  myop<<<1,size>>>(d_output,d_input,size);
  hipMemcpy(result,d_output,(size*8)*sizeof(float),hipMemcpyDeviceToHost);
  // for(int i = 0; i< 6*size; i++)
  // {
  //   printf("%f \t",result[i]);
  //   if(i%6==0)
  //   {
  //     printf("\n");
  //   }
  // }
  free(input);
  return result;

}
