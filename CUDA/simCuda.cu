
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#define fg 0.1
#define fe 100
#define damp 0.01
#define critical_factor  0.008
#define SCREEN_WIDTH 600
#define SCREEN_HEIGHT 600
const int MAX_SIZE = 500;
// [self.x,self.y,self.z,self.vx,self.vy,self.vz,self.m,self.charge]
//    0     1       2       3       4       5     6         7
__global__ void myop( float * d_output, float *d_input,int size)
{
  __shared__ float buffer[MAX_SIZE*8];
  int id = threadIdx.x;
  float my[8];
  for(int i = 0; i< 8; i++)
  {
    float temp  =d_input[8*id+i];
  buffer[8*id+i]=temp;
  my[i]=temp;

  }
  __syncthreads();

  for(int i = 0; i< size; i++)
  {
    if(i==id)
    continue;
    float other[8];
    for(int j = 0; j<8;j++)
    {
      other[j]=buffer[8*i+j];
    }
    float dx = my[0]-other[0];
    float dy = my[1]-other[1];
    float dz = my[2]-other[2];
    float r = sqrtf(dx*dx+dy*dy+dz*dz);
    if(r<3)
    {
      continue;
    }
    float fx =-(fg*my[6]*other[6]-fe*my[7]*other[7])*dx;
    fx=fx/(r*r*r);
    float fy =-(fg*my[6]*other[6]-fe*my[7]*other[7])*dy;
    fy=fy/(r*r*r);
    float fz =-(fg*my[6]*other[6]-fe*my[7]*other[7])*dz;
    fz=fz/(r*r*r);
    if(fx!=fx || fy!=fy||isnan(fy))
    continue;
    if(r>10)
    {
      my[3]+=(fx/my[6]);
      my[4]+=(fy/my[6]);
      my[5]+=(fz/my[6]);
    }
    else if(r>3)
    {
      my[3]-=fx*critical_factor*r*r;
      my[4]-=fy*critical_factor*r*r;
      my[5]-=fz*critical_factor*r*r;


      my[3]-=damp*(my[3]-other[3])*abs(my[2])*abs(my[7]-other[7])*abs(my[0]-other[0]);
      my[4]-=damp*(my[4]-other[4])*abs(my[3])*abs(my[7]-other[7])*abs(my[1]-other[1]);
      my[5]-=damp*(my[5]-other[5])*abs(my[3])*abs(my[7]-other[7])*abs(my[2]-other[2]);
    }

  }
  buffer[8*id+3]=my[3];
  buffer[8*id+4]=my[4];
  buffer[8*id+5]=my[5];
  buffer[8*id]+=my[3];
  buffer[8*id+1]+=my[4];
  buffer[8*id+2]+=my[5];
  __syncthreads();
  for(int i = 0; i< 8; i++)
  {
    d_output[8*id+i]=buffer[8*id+i];
  }
}
extern "C"{float * calcAndUpdate(float * input, int size)
{
  float * d_input, * d_output;
  float *result = (float *)malloc((size*6+2)*sizeof(float));
  if(size>500)
  {
    return 0;

  }
  hipMalloc(&d_input,(size*6)*sizeof(float));
  hipMalloc(&d_output,(size*6)*sizeof(float));
  hipMemcpy(d_input,input,(size*6)*sizeof(float),hipMemcpyHostToDevice);
  myop<<<1,size>>>(d_output,d_input,size);
  hipMemcpy(result,d_output,(size*6)*sizeof(float),hipMemcpyDeviceToHost);
  // for(int i = 0; i< 6*size; i++)
  // {
  //   printf("%f \t",result[i]);
  //   if(i%6==0)
  //   {
  //     printf("\n");
  //   }
  // }
  return result;

}}

int main()
{

}